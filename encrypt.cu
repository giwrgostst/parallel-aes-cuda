/******************************************************************************
 * AES-128 ENCRYPTION USING CUDA
 * 
 * Single-block demonstration:
 * - Encrypts one 128-bit block (plaintext) with a 128-bit key.
 ******************************************************************************/

#include <stdio.h>
#include <stdint.h>
#include <iostream>
#include <hip/hip_runtime.h>



// AES block size in bytes
#define AES_BLOCK_SIZE 16

// -----------------------------------------------------------------------------
// Device constants
// -----------------------------------------------------------------------------
__constant__ uint8_t d_sbox[256] = {
    0x63,0x7c,0x77,0x7b,0xf2,0x6b,0x6f,0xc5, 0x30,0x01,0x67,0x2b,0xfe,0xd7,0xab,0x76,
    0xca,0x82,0xc9,0x7d,0xfa,0x59,0x47,0xf0, 0xad,0xd4,0xa2,0xaf,0x9c,0xa4,0x72,0xc0,
    0xb7,0xfd,0x93,0x26,0x36,0x3f,0xf7,0xcc, 0x34,0xa5,0xe5,0xf1,0x71,0xd8,0x31,0x15,
    0x04,0xc7,0x23,0xc3,0x18,0x96,0x05,0x9a, 0x07,0x12,0x80,0xe2,0xeb,0x27,0xb2,0x75,
    0x09,0x83,0x2c,0x1a,0x1b,0x6e,0x5a,0xa0, 0x52,0x3b,0xd6,0xb3,0x29,0xe3,0x2f,0x84,
    0x53,0xd1,0x00,0xed,0x20,0xfc,0xb1,0x5b, 0x6a,0xcb,0xbe,0x39,0x4a,0x4c,0x58,0xcf,
    0xd0,0xef,0xaa,0xfb,0x43,0x4d,0x33,0x85, 0x45,0xf9,0x02,0x7f,0x50,0x3c,0x9f,0xa8,
    0x51,0xa3,0x40,0x8f,0x92,0x9d,0x38,0xf5, 0xbc,0xb6,0xda,0x21,0x10,0xff,0xf3,0xd2,
    0xcd,0x0c,0x13,0xec,0x5f,0x97,0x44,0x17, 0xc4,0xa7,0x7e,0x3d,0x64,0x5d,0x19,0x73,
    0x60,0x81,0x4f,0xdc,0x22,0x2a,0x90,0x88, 0x46,0xee,0xb8,0x14,0xde,0x5e,0x0b,0xdb,
    0xe0,0x32,0x3a,0x0a,0x49,0x06,0x24,0x5c, 0xc2,0xd3,0xac,0x62,0x91,0x95,0xe4,0x79,
    0xe7,0xc8,0x37,0x6d,0x8d,0xd5,0x4e,0xa9, 0x6c,0x56,0xf4,0xea,0x65,0x7a,0xae,0x08,
    0xba,0x78,0x25,0x2e,0x1c,0xa6,0xb4,0xc6, 0xe8,0xdd,0x74,0x1f,0x4b,0xbd,0x8b,0x8a,
    0x70,0x3e,0xb5,0x66,0x48,0x03,0xf6,0x0e, 0x61,0x35,0x57,0xb9,0x86,0xc1,0x1d,0x9e,
    0xe1,0xf8,0x98,0x11,0x69,0xd9,0x8e,0x94, 0x9b,0x1e,0x87,0xe9,0xce,0x55,0x28,0xdf,
    0x8c,0xa1,0x89,0x0d,0xbf,0xe6,0x42,0x68, 0x41,0x99,0x2d,0x0f,0xb0,0x54,0xbb,0x16
};

__constant__ uint8_t d_Rcon[11] = {
    0x00, // Rcon[0] is unused
    0x01, 0x02, 0x04, 0x08,
    0x10, 0x20, 0x40, 0x80,
    0x1B, 0x36
};

// -----------------------------------------------------------------------------
// Device utility functions
// -----------------------------------------------------------------------------
__device__ uint8_t galois_mul(uint8_t a, uint8_t b) {
    uint8_t p = 0;
    uint8_t hi_bit_set;
    for (int i = 0; i < 8; i++) {
        if (b & 1) p ^= a;
        hi_bit_set = (a & 0x80);
        a <<= 1;
        if (hi_bit_set) {
            // x^8 + x^4 + x^3 + x + 1 = 0x1B
            a ^= 0x1B;
        }
        b >>= 1;
    }
    return p;
}

__device__ void add_round_key(uint8_t state[4][4], uint8_t round_key[4][4]) {
    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            state[i][j] ^= round_key[i][j];
        }
    }
}

__device__ void sub_bytes(uint8_t state[4][4]) {
    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            state[i][j] = d_sbox[state[i][j]];
        }
    }
}

__device__ void shift_rows(uint8_t state[4][4]) {
    uint8_t temp;

    // Row 1: shift left by 1
    temp = state[1][0];
    state[1][0] = state[1][1];
    state[1][1] = state[1][2];
    state[1][2] = state[1][3];
    state[1][3] = temp;

    // Row 2: shift left by 2
    temp = state[2][0];
    state[2][0] = state[2][2];
    state[2][2] = temp;
    temp = state[2][1];
    state[2][1] = state[2][3];
    state[2][3] = temp;

    // Row 3: shift left by 3 (equivalent to shift right by 1)
    temp = state[3][3];
    state[3][3] = state[3][2];
    state[3][2] = state[3][1];
    state[3][1] = state[3][0];
    state[3][0] = temp;
}

__device__ void mix_columns(uint8_t state[4][4]) {
    uint8_t tmp[4];
    for (int i = 0; i < 4; i++) {
        tmp[0] = galois_mul(state[0][i], 2) ^ galois_mul(state[1][i], 3) ^ state[2][i] ^ state[3][i];
        tmp[1] = state[0][i] ^ galois_mul(state[1][i], 2) ^ galois_mul(state[2][i], 3) ^ state[3][i];
        tmp[2] = state[0][i] ^ state[1][i] ^ galois_mul(state[2][i], 2) ^ galois_mul(state[3][i], 3);
        tmp[3] = galois_mul(state[0][i], 3) ^ state[1][i] ^ state[2][i] ^ galois_mul(state[3][i], 2);

        state[0][i] = tmp[0];
        state[1][i] = tmp[1];
        state[2][i] = tmp[2];
        state[3][i] = tmp[3];
    }
}

__device__ void key_expansion(uint8_t key[4][4], uint8_t round_keys[11][4][4]) {
    uint32_t w[44];
    // Copy the original key into the first 4 words
    for (int i = 0; i < 4; i++) {
        w[i] = ((uint32_t)key[0][i] << 24) |
               ((uint32_t)key[1][i] << 16) |
               ((uint32_t)key[2][i] << 8)  |
               ((uint32_t)key[3][i]);
    }

    // Expand the key
    for (int i = 4; i < 44; i++) {
        uint32_t temp = w[i - 1];
        if ((i % 4) == 0) {
            // RotWord
            temp = (temp << 8) | (temp >> 24);
            // SubWord
            temp = ((uint32_t)d_sbox[(temp >> 24) & 0xFF] << 24) |
                   ((uint32_t)d_sbox[(temp >> 16) & 0xFF] << 16) |
                   ((uint32_t)d_sbox[(temp >> 8)  & 0xFF] << 8)  |
                   (uint32_t)d_sbox[temp & 0xFF];
            // Rcon
            temp ^= ((uint32_t)d_Rcon[i / 4] << 24);
        }
        w[i] = w[i - 4] ^ temp;
    }

    // Store expanded keys in round_keys
    for (int round = 0; round < 11; round++) {
        for (int i = 0; i < 4; i++) {
            uint32_t word = w[round * 4 + i];
            round_keys[round][0][i] = (word >> 24) & 0xFF;
            round_keys[round][1][i] = (word >> 16) & 0xFF;
            round_keys[round][2][i] = (word >> 8) & 0xFF;
            round_keys[round][3][i] = word & 0xFF;
        }
    }
}

__device__ void aes_encrypt_device(const uint32_t input[4], const uint32_t key[4], uint32_t output[4]) {
    // Convert input into state
    uint8_t state[4][4];
    for (int i = 0; i < 4; i++) {
        state[0][i] = (input[i] >> 24) & 0xFF;
        state[1][i] = (input[i] >> 16) & 0xFF;
        state[2][i] = (input[i] >> 8)  & 0xFF;
        state[3][i] =  input[i]        & 0xFF;
    }

    // Convert key into 4x4
    uint8_t key_state[4][4];
    for (int i = 0; i < 4; i++) {
        key_state[0][i] = (key[i] >> 24) & 0xFF;
        key_state[1][i] = (key[i] >> 16) & 0xFF;
        key_state[2][i] = (key[i] >> 8)  & 0xFF;
        key_state[3][i] =  key[i]        & 0xFF;
    }

    // Generate round keys
    uint8_t round_keys[11][4][4];
    key_expansion(key_state, round_keys);

    // Initial AddRoundKey
    add_round_key(state, round_keys[0]);

    // Rounds 1 to 9
    for (int round = 1; round <= 9; round++) {
        sub_bytes(state);
        shift_rows(state);
        mix_columns(state);
        add_round_key(state, round_keys[round]);
    }

    // Final Round
    sub_bytes(state);
    shift_rows(state);
    add_round_key(state, round_keys[10]);

    // Copy state back to output
    for (int i = 0; i < 4; i++) {
        output[i] = (state[0][i] << 24) |
                    (state[1][i] << 16) |
                    (state[2][i] << 8)  |
                     state[3][i];
    }
}

// -----------------------------------------------------------------------------
// CUDA Kernel
// -----------------------------------------------------------------------------
__global__ void aes_encrypt_kernel(const uint32_t *d_input, 
                                   const uint32_t *d_key, 
                                   uint32_t *d_output)
{
    // For a single block demonstration, we assume thread 0 does the work.
    // For multiple blocks, index with threadIdx, blockIdx.
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        aes_encrypt_device(d_input, d_key, d_output);
    }
}

int main() {
    // Example data
    uint32_t h_plaintext[4] = {
        0x3243f6a8, 
        0x885a308d, 
        0x313198a2, 
        0xe0370734
    };
    uint32_t h_key[4] = {
        0x2b7e1516, 
        0x28aed2a6, 
        0xabf71588, 
        0x09cf4f3c
    };
    uint32_t h_ciphertext[4]; // output
    uint32_t h_expected[4] = {
        0x3925841d, 
        0x02dc09fb, 
        0xdc118597, 
        0x196a0b32
    };

    // Allocate device memory
    uint32_t *d_plaintext, *d_key, *d_ciphertext;
    hipMalloc((void**)&d_plaintext,   4 * sizeof(uint32_t));
    hipMalloc((void**)&d_key,         4 * sizeof(uint32_t));
    hipMalloc((void**)&d_ciphertext,  4 * sizeof(uint32_t));

    // Copy data from host to device
    hipMemcpy(d_plaintext,  h_plaintext,  4 * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_key,        h_key,        4 * sizeof(uint32_t), hipMemcpyHostToDevice);

    // Launch kernel (1 block, 1 thread)
    aes_encrypt_kernel<<<1, 1>>>(d_plaintext, d_key, d_ciphertext);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(h_ciphertext, d_ciphertext, 4 * sizeof(uint32_t), hipMemcpyDeviceToHost);

    // Clean up device memory
    hipFree(d_plaintext);
    hipFree(d_key);
    hipFree(d_ciphertext);

    // Print results
    std::cout << "Encrypted ciphertext: ";
    for (int i = 0; i < 4; i++) {
        printf("%08x ", h_ciphertext[i]);
    }
    std::cout << std::endl;

    // Check
    bool match = true;
    for (int i = 0; i < 4; i++) {
        if (h_ciphertext[i] != h_expected[i]) {
            match = false;
            break;
        }
    }
    if (match) {
        std::cout << "Test Passed: The ciphertext matches the expected value!" << std::endl;
    } else {
        std::cout << "Test Failed: The ciphertext does NOT match the expected value." << std::endl;
    }

    return 0;
}
